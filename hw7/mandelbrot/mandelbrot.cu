#include "hip/hip_runtime.h"
/*  Based on fractal code by Martin Burtscher. */
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
extern "C" {
#include "anim.h"
}
const double Delta = 0.001;
const double xMid =  0.2370104;
const double yMid =  0.5210007;
int width;

static void quit() {
  printf("Usage: mandelbrot.exec WIDTH NSTEP FILENAME            \n\
  WIDTH = frame width, in pixels, (at least 10)                  \n\
  NSTEP = number of frames in the animation (at least 1)         \n\
  FILENAME = name of output file (to be created)                 \n\
Example: mandelbrot.exec 200 100 out.anim                        \n");
  exit(1);
}

__global__ void kernel(double * buf, double * xMin, double * yMin, double * dw){
   // Each block of size width.y gets width.x threads
   const int ltid = threadIdx.x; // local Thread ID
   const int btid = blockIdx.x; // block ID

   int i = ltid;
   const double cx = *xMin + i * *dw;
   int j = btid;
     const double cy = *yMin + j * *dw;
     double x = cx, y = cy, x2, y2;
     int depth = 256;

     do {
	x2 = x * x;
	y2 = y * y;
	y = 2 * x * y + cy;
	x = x2 - y2 + cx;
	depth--;
  } while (depth > 0 && x2 + y2 < 5.1);
  buf[i * blockDim.x + j] = (double)depth;
  __syncthreads();
}


int main(int argc, char *argv[]) {
  if (argc != 4) quit();
  // Setup Variables
  double start_time = ANIM_time();
  int dots = 0, nstep = atoi(argv[2]);
  width = atoi(argv[1]);
  char * filename = argv[3];
  int nblocks = width;   

  if (nstep < 1) quit();
  printf("mandelbrot: creating ANIM file %s with %d frames, %dx%d pixels, %zu bytes.\n",
	 filename, nstep, width, width,
	 ANIM_Heat_file_size_2d(width, width, nstep));

  ANIM_File af =
    ANIM_Create_heat_2d(width, width, 0, width, 0, width, 0, 255, filename);
  double * buf = (double*)malloc(width * width * sizeof(double)), delta = Delta;
  assert(buf);
  // Set Cuda Variables
  int err;
  double *dev_xMin;
  double *dev_yMin;
  double *dev_dw;
  int threadsPerBlock = width;
  double * dev_buf;
  err = hipMalloc((void**)&dev_buf, width*width*sizeof(double)); assert(err == hipSuccess);
err = hipMemcpy(dev_buf, buf, width*width*sizeof(double), hipMemcpyHostToDevice); assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_xMin, sizeof(double)); assert(err == hipSuccess); assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_yMin, sizeof(double)); assert(err == hipSuccess); assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_dw, sizeof(double)); assert(err == hipSuccess); assert(err == hipSuccess);
// Begin work
  for (int frame = 0; frame < nstep; frame++) {
    const double xMin = xMid - delta, yMin = yMid - delta;
    const double dw = 2.0 * delta / width;
    // send updated variables to kernel each iteration
    err = hipMemcpy(dev_xMin, &xMin, sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_yMin, &yMin, sizeof(double), hipMemcpyHostToDevice);
    err = hipMemcpy(dev_dw, &dw, sizeof(double), hipMemcpyHostToDevice);
    kernel<<<nblocks, threadsPerBlock>>>(dev_buf, dev_xMin, dev_yMin, dev_dw);
    // update host buffer
    err = hipMemcpy(buf, dev_buf, width*width*sizeof(double), hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    // update frame
    ANIM_Write_frame(af, buf);
    ANIM_Status_update(stdout, nstep, frame+1, &dots);
    delta *= 0.99;
  }
  ANIM_Close(af);
  printf("\nmandelbrot: finished.  Time = %lf\n", ANIM_time() - start_time);
  free(buf);
  hipFree(dev_buf);
  hipFree(dev_xMin);
  hipFree(dev_yMin);
  hipFree(dev_dw);
}
