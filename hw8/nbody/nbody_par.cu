#include "hip/hip_runtime.h"
/* nbody.cu: parallel 2-d nbody simulation
   Author: Stephen Siegel

   Link this with a translation unit that defines the extern
   variables, and anim.o, to make a complete program.
 */
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "nbody.h"
extern "C" {
#include "anim.h"
}
/* Global variables */
extern const double x_min;     /* coord of left edge of window (meters) */
extern const double x_max;     /* coord of right edge of window (meters) */
extern const double y_min;     /* coord of bottom edge of window (meters) */
extern const double y_max;     /* coord of top edge of window (meters) */
extern const int nx;           /* width of movie window (pixels) */
extern const int nbodies;      /* number of bodies */
extern const double delta_t;   /* time between discrete time steps (seconds) */
extern const int nstep;        /* number of time steps */
extern const int wstep;        /* number of times steps beween movie frames */
extern const int ncolors;      /* number of colors to use for the bodies */
extern const int colors[][3];  /* colors we will use for the bodies */
extern const Body bodies[];    /* list of bodies with initial data */
const double G = 6.674e-11;    /* universal gravitational constant */
int ny;                        /* height of movie window (pixels) */
State * states, * states_new;  /* two copies of state array */
ANIM_File af;                  /* output anim file */
double * posbuf;               /* to send data to anim, 2*nbodies doubles */
double start_time;             /* time simulation starts */

// Cuda Variable Declaration
int err;
int *dev_nbodies;
Body *dev_bodies;
State * dev_states, * dev_states_new;
double *dev_delta_t;
double *dev_x_min, *dev_x_max, *dev_y_min, *dev_y_max;

__device__ static inline double wrap(double x, const double min,
			  const double max, const double span) {
  while (x<min) x+=span;
  while (x>max) x-=span;
  return x;
}

static void init(char* filename) {
  #pragma omp parallel sections default(shared)
  {
    #pragma omp section
    {
      start_time = ANIM_time();
      //assert(x_max > x_min && y_max > y_min);
      ny = ceil(nx*(y_max - y_min)/(x_max - x_min));
      // Removed x & y span's here, as they can simply be calculated in the kernel
      printf("nbody: nbodies=%d nx=%d ny=%d nstep=%d wstep=%d\n",
      nbodies, nx, ny, nstep, wstep);
      const int nframes =  wstep == 0 ? 0 : 1+nstep/wstep;
      printf("nbody: creating ANIM file %s with %d frames, %zu bytes.\n",
      filename, nframes,
      ANIM_Nbody_file_size_2d(nbodies, ncolors, nframes));
      fflush(stdout);
      //assert(nx >= 10 && ny >= 10);
      //assert(nstep >= 1 && wstep >= 0 && nbodies > 0);
      //assert(ncolors >= 1 && ncolors <= ANIM_MAXCOLORS);
   }
    #pragma omp section
    {
      states = (State *)malloc(nbodies * sizeof(State));
      //assert(states);
      states_new = (State *)malloc(nbodies * sizeof(State));
      //assert(states_new);
      posbuf = (double *)malloc(2 * nbodies * sizeof(double));
      //assert(posbuf);
    } 
  }
  int radii[nbodies], bcolors[nbodies];
  ANIM_color_t acolors[ncolors]; // RGB colors converted to ANIM colors
  #pragma omp parallel
  #pragma omp for nowait
  for (int i=0; i<nbodies; i++) {
    //assert(bodies[i].mass > 0);
    //assert(bodies[i].color >= 0 && bodies[i].color < ncolors);
    //assert(bodies[i].radius > 0);
    states[i] = bodies[i].state;
    radii[i] = bodies[i].radius;
    bcolors[i] = bodies[i].color;
  }
  #pragma omp parallel for
  for (int i=0; i<ncolors; i++)
    acolors[i] = ANIM_Make_color(colors[i][0], colors[i][1], colors[i][2]);
  af =
    ANIM_Create_nbody_2d
    (nx, ny, x_min, x_max, y_min, y_max,
     nbodies, radii, ncolors, acolors, bcolors, filename);
  // Cuda Initilization & copying
  err = hipMalloc((void**)&dev_states, nbodies*sizeof(State)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_states, states, nbodies*sizeof(State), hipMemcpyHostToDevice); //assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_states_new, nbodies*sizeof(State)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_states_new, states_new, nbodies*sizeof(State), hipMemcpyHostToDevice); //assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_nbodies, sizeof(int)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_nbodies, &nbodies, sizeof(int), hipMemcpyHostToDevice); //assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_bodies, nbodies*sizeof(Body)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_bodies, bodies, nbodies*sizeof(Body), hipMemcpyHostToDevice); //assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_delta_t, sizeof(double)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_delta_t, &delta_t, sizeof(double), hipMemcpyHostToDevice); //assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_x_min, sizeof(double)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_x_min, &x_min, sizeof(double), hipMemcpyHostToDevice); //assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_x_max, sizeof(double)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_x_max, &x_max, sizeof(double), hipMemcpyHostToDevice); //assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_y_min, sizeof(double)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_y_min, &y_min, sizeof(double), hipMemcpyHostToDevice); //assert(err == hipSuccess);
  err = hipMalloc((void**)&dev_y_max, sizeof(double)); //assert(err == hipSuccess);
  err = hipMemcpy(dev_y_max, &y_max, sizeof(double), hipMemcpyHostToDevice); //assert(err == hipSuccess);
}

static inline void write_frame() {
  // Could use a parallel for here, but from testing it does not change performance
  for (int i=0; i<nbodies; i++) {
    posbuf[i*2] = states[i].x;
    posbuf[(i*2)+1] = states[i].y;
  }
  ANIM_Write_frame(af, posbuf);
}

/* Move forward one time step.  This is the "integration step".  For
   each body b, compute the total force acting on that body.  If you
   divide this by the mass of b, you get b's acceleration.  So you
   actually just calculate b's acceleration directly, since this is
   what you want to know.  Once you have the acceleration, update the
   velocity, then update the position. */

__global__ void update (State * states, State * states_new, int * nbodies, Body * bodies, double * delta_t, double * x_min, double * x_max, double * y_min, double * y_max){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    double x = states[tid].x, y = states[tid].y;
    double vx = states[tid].vx, vy = states[tid].vy;
    // ax times delta t, ay times delta t...
    double ax_delta_t = 0.0, ay_delta_t = 0.0;
    // Each thread calculates this section nbodies # of times per nstep
    for (int j=0; j<*nbodies; j++) {
      if (j == tid) continue;
      
      const double dx = states[j].x - x, dy = states[j].y - y;
      const double mass = bodies[j].mass;
      const double r_squared = dx*dx + dy*dy;
      
      if (r_squared != 0) {
	      const double r = sqrt(r_squared);
	
        if (r != 0) {
          const double acceleration = G * mass / r_squared;
          const double atOverr = acceleration * *delta_t / r;
          
          ax_delta_t += dx * atOverr;
          ay_delta_t += dy * atOverr;
        }
      }
    }
    vx += ax_delta_t;
    vy += ay_delta_t;
    x += *delta_t * vx;
    y += *delta_t * vy;
    //assert(!isnan(x) && !isnan(y) && !isnan(vx) && !isnan(vy));
    x = wrap(x, *x_min, *x_max, *x_max - *x_min);
    y = wrap(y, *y_min, *y_max, *y_max - *y_min);
    states_new[tid] = (State){x, y, vx, vy};
}

/* Close GIF file, free all allocated data structures */
static void wrapup() {
  ANIM_Close(af);
  free(posbuf);
  free(states);
  free(states_new);
  hipFree(dev_states);
  hipFree(dev_states_new);
  hipFree(dev_bodies);
  hipFree(dev_delta_t);
  hipFree(dev_nbodies);
  hipFree(dev_x_max);
  hipFree(dev_x_min);
  hipFree(dev_y_max);
  hipFree(dev_y_min);
  printf("\nnbody: finished.  Time = %lf\n", ANIM_time() - start_time);
}

/* Two arguments: the name of the output file and num. threads */
int main(int argc, char * argv[]) {
  int statbar = 0; // used for printing status updates

  assert(argc == 3);
  init(argv[2]);
  int threadsPerBlock;
  int nBlocks;
  if(nbodies >= 1001){
  threadsPerBlock = 1001;
  nBlocks = nbodies/threadsPerBlock;
  }
  else{
    threadsPerBlock = nbodies;
    nBlocks = 1;
  }
  if (wstep != 0) write_frame();
  
  for (int i=1; i<=nstep; i++) {
    update<<<nBlocks, threadsPerBlock>>>(dev_states, dev_states_new, dev_nbodies, dev_bodies, dev_delta_t, dev_x_min, dev_x_max, dev_y_min, dev_y_max);
    hipDeviceSynchronize();
    // Gather states & states_new from device
    err = hipMemcpy(states, dev_states, nbodies*sizeof(State), hipMemcpyDeviceToHost); //assert(err == hipSuccess);
    err = hipMemcpy(states_new, dev_states_new, nbodies*sizeof(State), hipMemcpyDeviceToHost); //assert(err == hipSuccess);
    // Pointer Swap
    State * const tmp = states; states = states_new; states_new = tmp;
    // Transfer swapped states back to device
    // This must be done sequentially, as there is no way to synchronize between blocks 
    // (if blocksize == 1 could be done in kernel, but negligible performance impact)
    err = hipMemcpy(dev_states, states, nbodies*sizeof(State), hipMemcpyHostToDevice); //assert(err == hipSuccess);
    err = hipMemcpy(dev_states_new, states_new, nbodies*sizeof(State), hipMemcpyHostToDevice); //assert(err == hipSuccess);
    ANIM_Status_update(stdout, nstep, i, &statbar);
    if (wstep != 0 && i%wstep == 0) write_frame();
  }
  wrapup();
}
